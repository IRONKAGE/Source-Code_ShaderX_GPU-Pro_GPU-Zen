#include "hip/hip_runtime.h"
//
// by Jan Eric Kyprianidis <www.kyprianidis.com>
// Copyright (C) 2010-2012 Computer Graphics Systems Group at the
// Hasso-Plattner-Institut, Potsdam, Germany <www.hpi3d.de>
//
// Permission to use, copy, modify, and/or distribute this software for any
// purpose with or without fee is hereby granted, provided that the above
// copyright notice and this permission notice appear in all copies.
// 
// THE SOFTWARE IS PROVIDED "AS IS" AND THE AUTHOR DISCLAIMS ALL WARRANTIES
// WITH REGARD TO THIS SOFTWARE INCLUDING ALL IMPLIED WARRANTIES OF
// MERCHANTABILITY AND FITNESS. IN NO EVENT SHALL THE AUTHOR BE LIABLE FOR
// ANY SPECIAL, DIRECT, INDIRECT, OR CONSEQUENTIAL DAMAGES OR ANY DAMAGES
// WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS, WHETHER IN AN
// ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS ACTION, ARISING OUT OF
// OR IN CONNECTION WITH THE USE OR PERFORMANCE OF THIS SOFTWARE.
//
#include "gpu_convert.h"


__global__ void imp_8u_to_32f( const gpu_plm2<unsigned char> src, gpu_plm2<float> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    unsigned char c = src(ix, iy);
    dst(ix, iy) = c / 255.0f;
}                       


__global__ void imp_8u_to_32f( const gpu_plm2<uchar4> src, gpu_plm2<float4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    uchar4 c = src(ix, iy);
    dst(ix, iy) = make_float4(c.x / 255.0f, c.y / 255.0f, c.z / 255.0f, c.w / 255.0f);
}                       


__global__ void imp_32f_to_8u( const gpu_plm2<float> src, gpu_plm2<unsigned char> dst) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    float c = clamp(src(ix, iy), 0.0f, 1.0f);
    dst(ix, iy) = (unsigned char)(255.0f *c);
}                       


__global__ void imp_32f_to_8u( const gpu_plm2<float4> src, gpu_plm2<uchar4> dst ) {
    const int ix = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
    const int iy = __mul24(blockDim.y, blockIdx.y) + threadIdx.y;
    if (ix >= dst.w || iy >= dst.h)
        return;

    float4 c = clamp(src(ix, iy), 0, 1);
    dst(ix, iy) = make_uchar4((int)(255.0f *c.x), (int)(255.0f *c.y), (int)(255.0f *c.z), (int)(255.0f *c.w));
}                       


gpu_image<float> gpu_8u_to_32f( const gpu_image<unsigned char>& src ) {
    gpu_image<float> dst(src.size());
    imp_8u_to_32f<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<float4> gpu_8u_to_32f( const gpu_image<uchar4>& src ) {
    gpu_image<float4> dst(src.size());
    GPU_CHECK_ERROR();
    imp_8u_to_32f<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<unsigned char> gpu_32f_to_8u( const gpu_image<float>& src ) {
    gpu_image<unsigned char> dst(src.size());
    imp_32f_to_8u<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}


gpu_image<uchar4> gpu_32f_to_8u( const gpu_image<float4>& src ) {
    gpu_image<uchar4> dst(src.size());
    imp_32f_to_8u<<<dst.blocks(), dst.threads()>>>(src, dst);
    GPU_CHECK_ERROR();
    return dst;
}
